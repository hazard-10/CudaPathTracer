#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"
#include <iostream>

namespace StreamCompaction
{
    namespace Efficient
    {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer &timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */

        __global__ void kernUpSweep(int n, int d, int *dev_data)
        {
            int index = (blockIdx.x * blockDim.x) + threadIdx.x;
            if (index >= n)
                return;
            int offset = 1 << (d + 1);
            if ((index + 1) % offset == 0)
            {
                int prev = index - (1 << d);
                dev_data[index] += dev_data[prev];
            }
        }

        __global__ void kernDownSweep(int n, int d, int *dev_data)
        {
            int index = (blockIdx.x * blockDim.x) + threadIdx.x;
            if (index >= n)
                return;
            int offset = 1 << (d + 1);
            if ((index + 1) % offset == 0)
            {
                int prev = index - (1 << d);
                int temp = dev_data[prev];
                dev_data[prev] = dev_data[index];
                dev_data[index] += temp;
            }
        }

        __global__ void setIndexAsZero(int n, int *odata)
        {
            odata[n] = 0;
        }

        void scan(int n, int *odata, const int *idata)
        {
            // TODO
            int blockSize = 256;
            dim3 fullBlocksPerGrid((blockSize + n - 1) / blockSize);

            int num_levels = ilog2ceil(n);
            int cudaArraySize = 1 << num_levels;
            int *dev_data;

            hipMalloc((void **)&dev_data, cudaArraySize * sizeof(int));
            hipMemset(dev_data, 0, n * sizeof(int));                             // initialize to 0
            hipMemcpy(dev_data, idata, n * sizeof(int), hipMemcpyHostToDevice); // copy data to device

            timer().startGpuTimer();
            for (int d = 0; d < num_levels; d++)
            {
                // std::cout << "up sweep d: " << d << std::endl;
                // build sum in place up the tree
                kernUpSweep<<<fullBlocksPerGrid, blockSize>>>(cudaArraySize, d, dev_data);
                // int *debug = new int[n];
                // hipMemcpy(debug, dev_data, n * sizeof(int), hipMemcpyDeviceToHost);
                // for (int i = 0; i < n; i++)
                // {
                //     std::cout << debug[i] << " ";
                // }
                // std::cout << std::endl;

                hipDeviceSynchronize();
            }
            setIndexAsZero<<<1, 1>>>(cudaArraySize - 1, dev_data);
            for (int d = num_levels - 1; d >= 0; d--)
            {
                // std::cout << "down sweep d: " << d << std::endl;
                kernDownSweep<<<fullBlocksPerGrid, blockSize>>>(cudaArraySize, d, dev_data);
                // int *debug = new int[n];
                // hipMemcpy(debug, dev_data, n * sizeof(int), hipMemcpyDeviceToHost);
                // for (int i = 0; i < n; i++)
                // {
                //     std::cout << debug[i] << " ";
                // }
                // std::cout << std::endl;

                hipDeviceSynchronize();
            }
            // TODO
            timer().endGpuTimer();
            hipMemcpy(odata, dev_data, n * sizeof(int), hipMemcpyDeviceToHost);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata)
        {
            // TODO

            // scan - initialize
            int blockSize = 256;
            dim3 fullBlocksPerGrid((blockSize + n - 1) / blockSize);
            int num_levels = ilog2ceil(n);
            int cudaArraySize = 1 << num_levels;
            int count = 0;
            int *dev_count;
            int *dev_data;    // holder for scan result
            int *dev_bools;   // holder for boolen array
            int *dev_inputs;  // copy of idata
            int *dev_outputs; // copy of odata
            hipMalloc((void **)&dev_count, sizeof(int));
            hipMalloc((void **)&dev_data, cudaArraySize * sizeof(int));
            hipMalloc((void **)&dev_bools, cudaArraySize * sizeof(int));
            hipMalloc((void **)&dev_inputs, n * sizeof(int));
            hipMalloc((void **)&dev_outputs, n * sizeof(int));
            hipMemset(dev_count, 0, sizeof(int));                                   // initialize to 0, size is 2^ilog2ceil(n
            hipMemset(dev_data, 0, cudaArraySize * sizeof(int));                    // initialize to 0, size is 2^ilog2ceil(n)
            hipMemset(dev_bools, 0, cudaArraySize * sizeof(int));                   // initialize to 0, size is 2^ilog2ceil(n)
            hipMemcpy(dev_inputs, idata, n * sizeof(int), hipMemcpyHostToDevice);  // copy data to device
            hipMemcpy(dev_outputs, odata, n * sizeof(int), hipMemcpyHostToDevice); // copy data to device

            // scan - prepare boolen array, get count
            StreamCompaction::Common::kernMapToBoolean<<<fullBlocksPerGrid, blockSize>>>(n, dev_bools, dev_inputs, dev_count);

            hipMemcpy(&count, dev_count, sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(dev_data, dev_bools, cudaArraySize * sizeof(int), hipMemcpyDeviceToDevice);

            // scan - scan boolean array
            timer().startGpuTimer();
            for (int d = 0; d < num_levels; d++)
            {
                // build sum in place up the tree
                kernUpSweep<<<fullBlocksPerGrid, blockSize>>>(cudaArraySize, d, dev_data);
                hipDeviceSynchronize();
            }
            setIndexAsZero<<<1, 1>>>(cudaArraySize - 1, dev_data);
            for (int d = num_levels - 1; d >= 0; d--)
            {
                kernDownSweep<<<fullBlocksPerGrid, blockSize>>>(cudaArraySize, d, dev_data);

                hipDeviceSynchronize();
            }
            // scan finished, next scatter
            StreamCompaction::Common::kernScatter<<<fullBlocksPerGrid,
                                                    blockSize>>>(n, dev_outputs, dev_inputs, dev_bools, dev_data);

            timer().endGpuTimer();

            hipMemcpy(odata, dev_outputs, count * sizeof(int), hipMemcpyDeviceToHost);

            return count;
        }
    }
}
